
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    // Vector size
    int numElements = 10;
    size_t size = numElements * sizeof(float);
    
    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize vectors on host
    for (int i = 0; i < numElements; i++) {
        h_A[i] = i;
        h_B[i] = i * 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy vectors from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Check for kernel errors
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print result vector
    printf("Vector A: ");
    for (int i = 0; i < numElements; i++) {
        printf("%.1f ", h_A[i]);
    }
    printf("\n");

    printf("Vector B: ");
    for (int i = 0; i < numElements; i++) {
        printf("%.1f ", h_B[i]);
    }
    printf("\n");

    printf("Result C (A+B): ");
    for (int i = 0; i < numElements; i++) {
        printf("%.1f ", h_C[i]);
    }
    printf("\n");

Error:
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset device
    hipDeviceReset();

    return 0;
} 